#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iterator>
#include <limits>
#include <list>
#include <vector>
#include <type_traits>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

template <typename T>
void print_2x2_matrix_values(T M, int ldM, std::string M_name)
{
  std::cout << std::endl;
  std::cout << "\t\t\t" << M_name << " = [ " << (float)M[0*ldM + 0] << ", " << (float)M[1*ldM + 0]         << ", ...\n";
  std::cout << "\t\t\t    [ "                << (float)M[0*ldM + 1] << ", " << (float)M[1*ldM + 1] << ", ...\n";
  std::cout << "\t\t\t    [ "                << "...\n";
  std::cout << std::endl;
}

//
// helpers for initializing templated scalar data type values.
//
template <typename fp> void rand_matrix(fp *M, int n_row, int n_col)
{
  for (int i = 0; i < n_row; i++)
    for (int j = 0; j < n_col; j++)
      M[i * n_col + j] = rand() % 2;
}

//
// Main example for Gemm consisting of
// initialization of A, B and C matrices as well as
// scalars alpha and beta.  Then the product
//
// C = alpha * op(A) * op(B) + beta * C
//
// is performed and finally the results are post processed.
//
template <typename fp>
void run_gemm_example(int m, int k, int n, int repeat) {

  //
  // Initialize data for Gemm
  //
  // C = alpha * op(A) * op(B)  + beta * C
  //

  // set scalar fp values
  const fp alpha = fp(2.0);
  const fp beta  = fp(0.5);

  const size_t A_size = sizeof(fp) * m * k;
  const size_t B_size = sizeof(fp) * k * n;
  const size_t C_size = sizeof(fp) * m * n;

  // prepare matrix data
  fp* a = (fp *) aligned_alloc(64, A_size);
  fp* b = (fp *) aligned_alloc(64, B_size);
  fp* c = (fp *) aligned_alloc(64, C_size);

  srand(2);
  rand_matrix(a, m, k);
  rand_matrix(b, k, n);
  rand_matrix(c, m, n);

  fp *da, *db, *dc;
  hipMalloc((void**)&da, A_size);
  hipMalloc((void**)&db, B_size);
  hipMalloc((void**)&dc, C_size);
  hipMemcpy(da, a, A_size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, B_size, hipMemcpyHostToDevice);

  // create execution queue and buffers of matrix data
  hipblasHandle_t h;
  hipblasCreate(&h);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    if constexpr (std::is_same_v<fp, __half>)
      hipblasHgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
    else if constexpr (std::is_same_v<fp, float>)
      hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
    else if constexpr (std::is_same_v<fp, double>)
      hipblasDgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                  &alpha, db, n, da, k, &beta, dc, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average GEMM execution time: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(c, dc, C_size, hipMemcpyDeviceToHost);
  hipblasDestroy(h);

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  //
  // Post Processing
  //

  std::cout << "\n\t\tOutputting 2x2 block of A,B,C matrices:" << std::endl;

  // output the top 2x2 block of A matrix
  //print_2x2_matrix_values(a, k, "A");

  // output the top 2x2 block of B matrix
  //print_2x2_matrix_values(b, n, "B");

  // output the top 2x2 block of C matrix
  print_2x2_matrix_values(c, n, "C");

  free(a);
  free(b);
  free(c);
}

//
// Main entry point for example.
//
int main (int argc, char ** argv) {
  if (argc != 5) {
    printf("Usage: %s <m> <k> <n> <repeat>\n", argv[0]);
    return 1;
  }
  const int m = atoi(argv[1]);
  const int k = atoi(argv[2]);
  const int n = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  std::cout << "\tRunning with half precision data type:" << std::endl;
  run_gemm_example<__half>(m, k, n, repeat);

  std::cout << "\tRunning with single precision data type:" << std::endl;
  run_gemm_example<float>(m, k, n, repeat);

  std::cout << "\tRunning with double precision data type:" << std::endl;
  run_gemm_example<double>(m, k, n, repeat);

  return 0;
}
